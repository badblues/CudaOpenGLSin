#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <glut.h>
#include <stdio.h>
#include <cmath>



int WinWid = 1280, WinHei = 720;  // Window width and height

int thread_number = 100;  // Number of threads/functions
int point_offset = 10, points_number = 50;  // Distance between points and number of points in one function
int y_offset = 2;  // Distance between functions, Y axis
int time_delay = 20;  // Frame update delay
int dx = 1;  // X change each frame

int* cpu_coordinates;  // Array of point coordinates
int* gpu_coordinates;  // Array of point coordinates, allocated on GPU
float* gpu_coefficients;  // Array of function coefficients, allocated on GPU


// Drawable function
__device__ int f(int x, float c1, float sin_coef, float cos_coef) {
    return int(100 * (sin_coef * sin(x/c1 * 3.14/180) + cos_coef * cos(x/c1 * 3.14/180)));
}

// Multithreading function, called each timer tick
__global__ void getNextPosition(int* coords, float* gpu_coefficients, int thread_num, int points_num, int dx, int y_offset) {
    int t_id = threadIdx.x;
	if(t_id < thread_num) {  // Overflow check
        int thread_offset = t_id * points_num * 2;  // Offset in array between threads
        // Getting values from coefficient array
        float c1 = gpu_coefficients[t_id * 3];
        float sin_coef = gpu_coefficients[t_id * 3 + 1];
        float cos_coef = gpu_coefficients[t_id * 3 + 2];
		for (int i = 0; i < points_num; i++)
        {
            // Calculating new points
            coords[thread_offset + i * 2] += dx;
            coords[thread_offset + i * 2 + 1] = f(coords[thread_offset + i * 2], c1, sin_coef, cos_coef) + t_id * y_offset - (thread_num * y_offset / 2);
        }
        
    }
}

// Redrawing function
void draw() {

    glClear(GL_COLOR_BUFFER_BIT);
    glEnable(GL_POINT_SMOOTH);
    glPushMatrix();
    glScalef(1 / ((float)WinWid / 2), 1 / ((float)WinHei / 2), 1);


    for (int i = 0; i < thread_number * points_number; i++) {
    	glColor3f(1, 0, 0);
    	glBegin(GL_POINTS);
        glPointSize(15);
        glVertex2i(cpu_coordinates[i*2] % WinWid - WinWid / 2, cpu_coordinates[i*2+1]);
        glEnd();
    }

    glPopMatrix();
    glutSwapBuffers();

}

// Timer function, called every time_delay msec
void timer(int value) {

    int size = thread_number * points_number * 2;
    hipMemcpy(gpu_coordinates, cpu_coordinates, size * sizeof(int), hipMemcpyHostToDevice);  // Copying from CPU
    getNextPosition<<<1, thread_number >>>(gpu_coordinates, gpu_coefficients, thread_number, points_number, dx, y_offset);  // Calculating next coordinates
    hipDeviceSynchronize();  // Synchronizing threads
    hipMemcpy(cpu_coordinates, gpu_coordinates, size * sizeof(int), hipMemcpyDeviceToHost);  // Copying back to CPU

    // Calling redrawing
    glutPostRedisplay();  
    glutTimerFunc(time_delay, timer, 0);

}


// Filling our thread/function coefficients with random numbers
void fillCoefficients() {
    int size = thread_number * 3;
    hipMalloc(&gpu_coefficients, size * sizeof(float));
    float* ptr = (float*)malloc(size * sizeof(float));
    for (int i = 0; i < thread_number; i++) {
        ptr[i * 3] = (float)rand() / (float)RAND_MAX * (5 - 2) + 1;  //[2, 5]
        ptr[i * 3 + 1] = (float)rand() / (float)RAND_MAX;  //[0, 1]
        ptr[i * 3 + 2] = (float)rand() / (float)RAND_MAX;  //[0, 1]
    }
    hipMemcpy(gpu_coefficients, ptr, size * sizeof(float), hipMemcpyHostToDevice);
}


// Initializing function
void init() {

    glClearColor(0.0, 0.0, 0.0, 1.0);

    glMatrixMode(GL_PROJECTION);

    srand(time(NULL));
    cpu_coordinates = (int*)malloc(thread_number * points_number * 2 * sizeof(int));
    for (int i = 0; i < thread_number; i++) {
        int thread_offset = i * points_number * 2;
        int random_x_offset = rand() % WinWid;
        for (int j = 0; j < points_number; j++)
            cpu_coordinates[thread_offset + j * 2] = random_x_offset + point_offset * j;
    }
    fillCoefficients();

    int size = thread_number * points_number * 2;
    hipMalloc(&gpu_coordinates, size * sizeof(int));  // Allocating memory for coordinates array

    glLoadIdentity();
    glMatrixMode(GL_MODELVIEW);

}





int main(int argc, char** argv) {

    // Setting up OpenGL window
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);


    glutInitWindowSize(WinWid, WinHei);
    glutInitWindowPosition(400, 150);
    glutCreateWindow("SIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIIINUS");


    glutDisplayFunc(draw);
    glutTimerFunc(60, timer, 0);

    init();

    glutMainLoop();

    // Releasing allocated memory
    free(cpu_coordinates);
    hipFree(gpu_coordinates);  
    hipFree(gpu_coefficients);

}